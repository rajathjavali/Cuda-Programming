
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<random>

#define cudaCheck(x) _cudaCheck(x, #x ,__FILE__, __LINE__)
#define BLOCKTHREAD 1024

template<typename T>
void _cudaCheck(T e, const char* func, const char* call, const int line){
  if(e != hipSuccess){
    printf("\"%s\" at %d in %s\n\treturned %d\n-> %s\n", func, line, call, (int)e, hipGetErrorString(e));
    exit(EXIT_FAILURE);
  }
}

// Number of elements worked by one thread
long nelem;

// TODO kernel code here: each threads work on nelem - elements in a pair of sz-long vector
__global__ void add_krnl(float *x, float *y, long sz, long nelem) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	int num_threads = (sz+nelem-1)/nelem;
	int j, ind;


	for (j = 0; j < nelem; j+=1) {
	  ind = j*num_threads + id; // cyclic
	 // ind = j + id*nelem; // adjacent
		if ((ind < sz) & (id < num_threads)) {
			x[ind] = x[ind] + y[ind];
		}
	}
}

long func_add(float *x, float *y, long sz) {
  long i;
  float *d_x_array, *d_y_array;
  int num_blocks;

  /* TODO: Make call to GPU kernel to compute results on GPU into d_x */
  /* STEP1: Write thread program to compute nelem elements per thread */
  /* STEP2: Invoke thread program for sz */
  /* STEP3: Using count6 as example, allocate GPU input/output data */
  /* STEP3: Using count6 as example, initialize GPU input/output data */
  
  hipMalloc((void **) &d_x_array,sz*sizeof(float));
  hipMalloc((void **) &d_y_array,sz*sizeof(float));
  hipMemcpy(d_x_array, x, sz*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_y_array, y, sz*sizeof(float),hipMemcpyHostToDevice);

  // Timing using cudaEvent
  hipEvent_t start, stop;
  float et;
  cudaCheck(hipEventCreate(&start));
  cudaCheck(hipEventCreate(&stop));

  num_blocks = (sz + BLOCKTHREAD - 1)/BLOCKTHREAD;

  // Time event start
  cudaCheck(hipEventRecord(start));
  
  {
    // TODO Invoke the kernel code here
    add_krnl <<<num_blocks, BLOCKTHREAD>>> (d_x_array, d_y_array, sz, nelem);
  }

  cudaCheck(hipGetLastError());

  // Time event end
  cudaCheck(hipEventRecord(stop));
  cudaCheck(hipEventSynchronize(stop));
  cudaCheck(hipEventElapsedTime(&et, start, stop));
  cudaCheck(hipEventDestroy(start));
  cudaCheck(hipEventDestroy(stop));

  printf("\t%0.3f", et);

  // TODO Copy data back to d_x and free GPU memory
  float * d_x = (float *) malloc(sz * sizeof(float));

  hipMemcpy(d_x, d_x_array, sz*sizeof(float),hipMemcpyDeviceToHost);

  /* CPU Calculation */
  for (i = 0; i < sz; i++) {
    x[i] += y[i];
  }

  /* Compare CPU and GPU output to see if it is within error tolerance */
  for (i = 0; i < sz; i++) {
    if (fabsf(d_x[i] - x[i]) > 1e-5) {
      hipFree(d_x_array);
      hipFree(d_y_array);
      free(d_x);
      return 0;
    }
  }
  hipFree(d_x_array);
  hipFree(d_y_array);
  free(d_x);
  return 1;
}


int main(int argc, char **argv) {
  float *a, *b;
  long j;
  long i;

  std::random_device rd;
  std::mt19937_64 mt(rd());
  std::uniform_real_distribution<float> u(0, 1);

  // Print title
  printf("sz");
  for (nelem = 1; nelem < 513; nelem *= 2)
    printf("\t%d", nelem);
  printf("\n");

  for (j = 10; j <= 1000000000; j *= 10) {
    a = (float *) malloc(sizeof(float) * j);
    b = (float *) malloc(sizeof(float) * j);

    /* Initialize with random number generator */
    for (i = 0; i < j; i++) {
      a[i] = u(mt);
      b[i] = u(mt);
    }

    printf("%d", j);

    for (nelem = 1; nelem < 513; nelem *= 2)
      if (!func_add(a, b, j))
        printf("failed to add\n");

    printf("\n");

    free(a);
    free(b);
  }

  return 0;
}
